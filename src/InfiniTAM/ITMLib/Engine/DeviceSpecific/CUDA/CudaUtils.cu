#include "CudaUtils.h"

#include "../../../Utils/ITMLibDefines.h"

bool getCudaMemoryUsage(size_t& free_byte, size_t& total_byte)
{
    return (hipMemGetInfo(&free_byte, &total_byte) == hipSuccess);
}