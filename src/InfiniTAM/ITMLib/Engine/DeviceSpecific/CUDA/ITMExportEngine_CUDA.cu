#include "hip/hip_runtime.h"

#include "ITMExportEngine_CUDA.h"

#include "../../../../ORUtils/CUDADefines.h"

#include <fstream>
#include <vector>

using namespace ITMLib::Engine;
using namespace ITMLib::Objects;


namespace ITMLib
{
    namespace Engine
    {
        template<class TVoxel>
        __global__ void getVoxelsFromPtrList_device(int noBlocks, int voxelsPerBlock,
                                                    const int* blockPtr,
                                                    const TVoxel* voxels_in, TVoxel* voxels_out)
        {
            int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
            if (targetIdx > noBlocks - 1) return;

            for(int i = 0; i < voxelsPerBlock; i++)
            {
                voxels_out[targetIdx * voxelsPerBlock + i] =
                        voxels_in[blockPtr[targetIdx] * voxelsPerBlock + i];
            }
        }
    }
}


namespace ITMLib
{
    namespace Engine
    {
        template<class TVoxel, class TIndex>
        int ITMExportEngine_CUDA<TVoxel, TIndex>::countVoxelsInBlock(const TVoxel* voxelBlocks,
                                                                     int block_ptr)
        {
            int noPoints = 0;

            for(int z = 0; z < SDF_BLOCK_SIZE; z++)
            {
                for(int y = 0; y < SDF_BLOCK_SIZE; y++)
                {
                    for(int x = 0; x < SDF_BLOCK_SIZE; x++)
                    {
                        int point_offset = x + y*SDF_BLOCK_SIZE + z*SDF_BLOCK_SIZE*SDF_BLOCK_SIZE;

                        const TVoxel& voxel = voxelBlocks[block_ptr * SDF_BLOCK_SIZE3 + point_offset];

                        if(voxel.sdf != TVoxel::SDF_initialValue()){ noPoints++; }
                    }
                }
            }

            return noPoints;
        }


        template<class TVoxel, class TIndex>
        void ITMExportEngine_CUDA<TVoxel, TIndex>::
        getVoxelsFromPtrList(int noBlocks,
                             const TVoxel* voxelBlocks_device,
                             int* blockPtr_host,
                             TVoxel* voxels_host)
        {
            TVoxel* voxels_device;
            int* blockPtr_device;

            ITMSafeCall(hipMalloc((void**)&voxels_device, noBlocks*SDF_BLOCK_SIZE3*sizeof(TVoxel)));
            ITMSafeCall(hipMalloc((void**)&blockPtr_device, noBlocks*sizeof(int)));

            ITMSafeCall(hipMemcpy(blockPtr_device, blockPtr_host, noBlocks*sizeof(int), hipMemcpyHostToDevice));

            dim3 cudaBlockSizeAL(256, 1);
            dim3 gridSizeAL((int)ceil((float)noBlocks / (float)cudaBlockSizeAL.x));

            getVoxelsFromPtrList_device<<<gridSizeAL, cudaBlockSizeAL>>>(
                    noBlocks, SDF_BLOCK_SIZE3, blockPtr_device,
                    voxelBlocks_device, voxels_device);

            ITMSafeCall(hipMemcpy(voxels_host, voxels_device, noBlocks*SDF_BLOCK_SIZE3*sizeof(TVoxel), hipMemcpyDeviceToHost));

            ITMSafeCall(hipFree(voxels_device));
            ITMSafeCall(hipFree(blockPtr_device));
        }

        template<class TVoxel, class TIndex>
        void ITMExportEngine_CUDA<TVoxel, TIndex>::exportVoxelsInBlock(std::ofstream& file,
                                                                       const TVoxel* voxelBlocks,
                                                                       int block_ptr,
                                                                       Vector3s blockPos)
        {
            Vector3f pointPos = blockPos.toFloat() * SDF_BLOCK_SIZE;

            for(int z = 0; z < SDF_BLOCK_SIZE; z++)
            {
                for(int y = 0; y < SDF_BLOCK_SIZE; y++)
                {
                    for(int x = 0; x < SDF_BLOCK_SIZE; x++)
                    {
                        int point_offset = x + y*SDF_BLOCK_SIZE + z*SDF_BLOCK_SIZE*SDF_BLOCK_SIZE;
                        int voxel_ptr = block_ptr * SDF_BLOCK_SIZE3 + point_offset;
                        const TVoxel& voxel = voxelBlocks[voxel_ptr];

                        if(voxel.sdf == TVoxel::SDF_initialValue()){ continue; }

                        // Export X,Y,Z
                        Vector3f currPointPos = pointPos + Vector3f(x, y, z);
                        //file << currPointPos.x << " " << currPointPos.y << " " << currPointPos.z << " ";
                        file.write((const char*)&currPointPos, 3 * sizeof(float));


                        // Export colors
                        if(TVoxel::hasColorInformation)
                        {
                            // hack to get color information without using templates
                            // TODO use templates
                            const uchar* raw_ptr = reinterpret_cast<const uchar*>(voxelBlocks);
                            const uchar* clr_ptr = raw_ptr + (voxel_ptr+1) * sizeof(TVoxel) -
                                    sizeof(uchar) - sizeof(Vector3u) - 1;

                            unsigned int rgb = ((unsigned int)clr_ptr[0] << 16 | (unsigned int)clr_ptr[1] << 8 | (unsigned int)clr_ptr[2]);
                            float rgb_f = *reinterpret_cast<float*>(&rgb);

                            //file << rgb_f << " ";
                            file.write((const char*)&rgb_f, sizeof(float));
                        }

                        // Export SDF value
                        float tsdf = TVoxel::SDF_valueToFloat(voxel.sdf);
                        //file << tsdf << "\n";
                        file.write((const char*)&tsdf, sizeof(float));
                    }
                }
            }
        }


        template<class TVoxel, class TIndex>
        bool ITMExportEngine_CUDA<TVoxel, TIndex>::ExportTSDFToPcd_hashIndex(
                Objects::ITMScene<TVoxel,
                ITMVoxelBlockHash>* scene,
                const char *filename)
        {
            if(!scene){ return false; }

            ITMGlobalCache<TVoxel>* cache = scene->globalCache;

            bool hasGlobalCache = scene->useSwapping;


            TVoxel* voxelBlocks_global = 0;
            TVoxel* voxelBlocks_local = scene->localVBA.GetVoxelBlocks();

            if(hasGlobalCache){ voxelBlocks_global = cache->GetStoredVoxelBlock(0); }

            ORUtils::MemoryBlock<ITMHashEntry> hashTable(1, MEMORYDEVICE_CPU);
            scene->index.getEntriesCPUCopy(hashTable);
            ITMHashEntry* hash = hashTable.GetData(MEMORYDEVICE_CPU);


            int noEntries = hashTable.dataSize;

            ITMHashSwapState swapStateDefault;
            swapStateDefault.state = 2;
            std::vector<ITMHashSwapState> swapStates(noEntries, swapStateDefault);

            if(hasGlobalCache)
            {
                ITMSafeCall(hipMemcpy(swapStates.data(), cache->GetSwapStates(true), noEntries*sizeof(ITMHashSwapState), hipMemcpyDeviceToHost));
            }

            const size_t MAX_BLOCKS_TRANSFER_SIZE = 512;

            int noPoints = 0;
            std::vector<int> localPtr;
            std::vector<TVoxel> localVoxels_host;
            for(int i = 0; i < noEntries; i++)
            {
                if(hash[i].ptr >= -1 && swapStates[i].state != 2)
                {
                    noPoints += countVoxelsInBlock(voxelBlocks_global, i);
                }
                else if(hash[i].ptr >= 0 && swapStates[i].state == 2)
                {
                    localPtr.push_back(hash[i].ptr);
                }

                if(localPtr.size() >= MAX_BLOCKS_TRANSFER_SIZE || (i >= noEntries-1 && !localPtr.empty()))
                {
                    localVoxels_host.resize(localPtr.size() * SDF_BLOCK_SIZE3);

                    getVoxelsFromPtrList(localPtr.size(), voxelBlocks_local, localPtr.data(), localVoxels_host.data());

                    for(size_t j = 0; j < localPtr.size(); j++)
                    {
                        noPoints += countVoxelsInBlock(localVoxels_host.data(), j);
                    }

                    localPtr.clear();
                }
            }


            if(noPoints <= 0){ return false; }

            std::ofstream file(filename);
            if(!file.is_open()){ return false; }

            file << "VERSION 0.7\n";
            file << "FIELDS x y z " << (TVoxel::hasColorInformation? "rgb": "") << " intensity\n";
            file << "SIZE 4 4 4 " << (TVoxel::hasColorInformation? "4": "") << " 4\n";
            file << "TYPE F F F " << (TVoxel::hasColorInformation? "F": "") << " F\n";
            file << "COUNT 1 1 1 " << (TVoxel::hasColorInformation? "1": "") << " 1\n";
            file << "WIDTH " << noPoints << "\n";
            file << "HEIGHT 1\n";
            file << "VIEWPOINT 0 0 0 1 0 0 0\n";
            file << "POINTS " << noPoints << "\n";
            file << "DATA binary\n";


            std::vector<Vector3s> localPtrPos;
            for(int i = 0; i < noEntries; i++)
            {
                if(hash[i].ptr >= -1 && swapStates[i].state != 2)
                {
                    exportVoxelsInBlock(file, voxelBlocks_global, i, hash[i].pos);
                }
                else if(hash[i].ptr >= 0 && swapStates[i].state == 2)
                {
                    localPtr.push_back(hash[i].ptr);
                    localPtrPos.push_back(hash[i].pos);
                }

                if(localPtr.size() >= MAX_BLOCKS_TRANSFER_SIZE || (i >= noEntries-1 && !localPtr.empty()))
                {
                    localVoxels_host.resize(localPtr.size() * SDF_BLOCK_SIZE3);

                    getVoxelsFromPtrList(localPtr.size(), voxelBlocks_local, localPtr.data(), localVoxels_host.data());

                    for(size_t j = 0; j < localPtr.size(); j++)
                    {
                        exportVoxelsInBlock(file, localVoxels_host.data(), j, localPtrPos[j]);
                    }

                    localPtr.clear();
                    localPtrPos.clear();
                }
            }

            return true;
        }


        template<class TVoxel, class TIndex>
        bool ITMExportEngine_CUDA<TVoxel, TIndex>::ExportTSDFToPcd(const char *filename)
        { return false; }


        template <>
        bool ITMExportEngine_CUDA<ITMVoxel_s_rgb, ITMVoxelBlockHash>::ExportTSDFToPcd(const char *filename)
        {
            return ExportTSDFToPcd_hashIndex(scene_, filename);
        }

        template <>
        bool ITMExportEngine_CUDA<ITMVoxel_s, ITMVoxelBlockHash>::ExportTSDFToPcd(const char *filename)
        {
            return ExportTSDFToPcd_hashIndex(scene_, filename);
        }

        template <>
        bool ITMExportEngine_CUDA<ITMVoxel_f, ITMVoxelBlockHash>::ExportTSDFToPcd(const char *filename)
        {
            return ExportTSDFToPcd_hashIndex(scene_, filename);
        }

        template <>
        bool ITMExportEngine_CUDA<ITMVoxel_f_rgb, ITMVoxelBlockHash>::ExportTSDFToPcd(const char *filename)
        {
            return ExportTSDFToPcd_hashIndex(scene_, filename);
        }



    } // namespace
} // namespace