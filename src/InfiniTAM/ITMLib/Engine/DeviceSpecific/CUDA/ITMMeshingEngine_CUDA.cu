#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMMeshingEngine_CUDA.h"
#include "../../DeviceAgnostic/ITMMeshingEngine.h"
#include "ITMCUDAUtils.h"

#include "../../../../ORUtils/CUDADefines.h"

template<class TVoxel>
__global__ void meshScene_device(ITMMesh::Triangle *triangles, unsigned int *noTriangles_device, float factor, int noTotalEntries,
	int noMaxTriangles, const Vector4s *visibleBlockGlobalPos, const TVoxel *localVBA, const ITMHashEntry *hashTable);

__global__ void findAllocateBlocks(Vector4s *visibleBlockGlobalPos, const ITMHashEntry *hashTable, int noTotalEntries);

using namespace ITMLib::Engine;

template<class TVoxel>
ITMMeshingEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMMeshingEngine_CUDA(void) 
{
	ITMSafeCall(hipMalloc((void**)&visibleBlockGlobalPos_device, SDF_LOCAL_BLOCK_NUM * sizeof(Vector4s)));
	ITMSafeCall(hipMalloc((void**)&noTriangles_device, sizeof(unsigned int)));
}

template<class TVoxel>
ITMMeshingEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMMeshingEngine_CUDA(void) 
{
	ITMSafeCall(hipFree(visibleBlockGlobalPos_device));
	ITMSafeCall(hipFree(noTriangles_device));
}

template<class TVoxel>
void ITMMeshingEngine_CUDA<TVoxel, ITMVoxelBlockHash>::MeshScene(ITMMesh *mesh, const ITMScene<TVoxel, ITMVoxelBlockHash> *scene)
{
	ITMMesh::Triangle *triangles = mesh->triangles->GetData(MEMORYDEVICE_CUDA);
	const TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	const ITMHashEntry *hashTable = scene->index.GetEntries();

	int noMaxTriangles = mesh->noMaxTriangles, noTotalEntries = scene->index.noTotalEntries;
	float factor = scene->sceneParams->voxelSize;

	ITMSafeCall(hipMemset(noTriangles_device, 0, sizeof(unsigned int)));
	ITMSafeCall(hipMemset(visibleBlockGlobalPos_device, 0, sizeof(Vector4s) * SDF_LOCAL_BLOCK_NUM));

	{ // identify used voxel blocks
		dim3 cudaBlockSize(256); 
		dim3 gridSize((int)ceil((float)noTotalEntries / (float)cudaBlockSize.x));

		findAllocateBlocks << <gridSize, cudaBlockSize >> >(visibleBlockGlobalPos_device, hashTable, noTotalEntries);
	}

	{ // mesh used voxel blocks
		dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
		dim3 gridSize(SDF_LOCAL_BLOCK_NUM / 16, 16);

		meshScene_device<TVoxel> << <gridSize, cudaBlockSize >> >(triangles, noTriangles_device, factor, noTotalEntries, noMaxTriangles,
			visibleBlockGlobalPos_device, localVBA, hashTable);

		ITMSafeCall(hipMemcpy(&mesh->noTotalTriangles, noTriangles_device, sizeof(unsigned int), hipMemcpyDeviceToHost));
	}
}

template<class TVoxel>
ITMMeshingEngine_CUDA<TVoxel,ITMPlainVoxelArray>::ITMMeshingEngine_CUDA(void) 
{}

template<class TVoxel>
ITMMeshingEngine_CUDA<TVoxel,ITMPlainVoxelArray>::~ITMMeshingEngine_CUDA(void) 
{}

template<class TVoxel>
void ITMMeshingEngine_CUDA<TVoxel, ITMPlainVoxelArray>::MeshScene(ITMMesh *mesh, const ITMScene<TVoxel, ITMPlainVoxelArray> *scene)
{}

__global__ void findAllocateBlocks(Vector4s *visibleBlockGlobalPos, const ITMHashEntry *hashTable, int noTotalEntries)
{
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId > noTotalEntries - 1) return;

	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr >= 0) 
		visibleBlockGlobalPos[currentHashEntry.ptr] = Vector4s(currentHashEntry.pos.x, currentHashEntry.pos.y, currentHashEntry.pos.z, 1);
}

template<class TVoxel>
__global__ void meshScene_device(ITMMesh::Triangle *triangles, unsigned int *noTriangles_device, float factor, int noTotalEntries, 
	int noMaxTriangles, const Vector4s *visibleBlockGlobalPos, const TVoxel *localVBA, const ITMHashEntry *hashTable)
{
	const Vector4s globalPos_4s = visibleBlockGlobalPos[blockIdx.x + gridDim.x * blockIdx.y];

	if (globalPos_4s.w == 0) return;

	Vector3i globalPos = Vector3i(globalPos_4s.x, globalPos_4s.y, globalPos_4s.z) * SDF_BLOCK_SIZE;

	Vector3f vertList[12];
	int cubeIndex = buildVertList(vertList, globalPos, Vector3i(threadIdx.x, threadIdx.y, threadIdx.z), localVBA, hashTable);

	if (cubeIndex < 0) return;

	for (int i = 0; triangleTable[cubeIndex][i] != -1; i += 3)
	{
		int triangleId = atomicAdd(noTriangles_device, 1);

		if (triangleId < noMaxTriangles - 1)
		{
			triangles[triangleId].p0 = vertList[triangleTable[cubeIndex][i]] * factor;
			triangles[triangleId].p1 = vertList[triangleTable[cubeIndex][i + 1]] * factor;
			triangles[triangleId].p2 = vertList[triangleTable[cubeIndex][i + 2]] * factor;
		}
	}
}

template class ITMLib::Engine::ITMMeshingEngine_CUDA<ITMVoxel, ITMVoxelIndex>;
